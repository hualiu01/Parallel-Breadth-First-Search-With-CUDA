// This is a parallel Breath First Search (BFS) Implementation using CUDA
// Input: randomly generated graph in form of adjacent matrix stored in 'mygraph.txt'
// BFS starts from node 0 and stops when the logical BFS tree is formed
// usage: ./<out> <graph_size> <mygraph.txt


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <time.h>

using namespace std;

__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier){
	int row_idx = frontier[blockIdx.x+1]; 
	long offset = N * row_idx;

	// update new_frontier in threads
	int col_idx = threadIdx.x;	
	if(adj_mat[offset + col_idx] && !visited[col_idx]){
		new_frontier[col_idx] = true;
	}
}
__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier){
	int cn = 0;
	for(int i=0;i<N;i++){
		if(new_frontier[i]){
			new_frontier[i] = false;
			frontier[++cn] = i;
			visited[i] = true;
		}
	}
	frontier[0] = cn;
}

int main(int arg, char** argv){
	if(arg!=2){
		cout<<"usage: ./<out> <graph_size> >mygraph.txt"<<endl;
		return -1;
	}
	const int N = atoi(argv[1]);

	//read graph from <input>.txt
	bool* h_adj_mat = (bool*)malloc(N*N*sizeof(bool));
	for(int i=0;i<N*N;i++){
		string a;
		cin>>a;
		if(a=="1") h_adj_mat[i] = true;
		else h_adj_mat[i] = false;		
	}

	//generate visited and frontier vector; init them with node 0;
	bool* h_visited = (bool*)malloc(N*sizeof(bool));
	for(int i=0;i<N;i++) h_visited[i] = false;
	int* h_frontier = (int*)malloc(N*sizeof(int));
	bool* h_new_frontier = (bool*)malloc(N*sizeof(bool));
	for(int i=0;i<N;i++) h_new_frontier[i] = false;

	h_visited[0] = true;
	h_frontier[0] = 1;
	h_frontier[1] = 0;
	
	//malloc mem in gpu
	clock_t start,end, s, e;
	start = clock();
	bool *d_adj_mat, *d_visited, *d_new_frontier;
	int *d_frontier;
	hipMalloc((void**) &d_adj_mat, sizeof(bool) * N * N);
	hipMemcpy((void*) d_adj_mat, (void*) h_adj_mat, sizeof(bool)*N*N, hipMemcpyHostToDevice);
	
	hipMalloc((void**) &d_visited, sizeof(bool) * N);
	hipMemcpy((void*) d_visited, (void*) h_visited, sizeof(bool)*N, hipMemcpyHostToDevice);
	
	hipMalloc((void**) &d_frontier, sizeof(int) * (N+1));
	hipMemcpy((void*) d_frontier, (void*) h_frontier, sizeof(int)*N, hipMemcpyHostToDevice);
	
	hipMalloc((void**) &d_new_frontier, sizeof(bool) * N);
	hipMemcpy((void*) d_new_frontier, (void*) h_new_frontier, sizeof(bool)*N, hipMemcpyHostToDevice);

	//loop until frontier vector is empty 
	int cn =1;
	double t=0;
	while(h_frontier[0]!=0){
		cn+=h_frontier[0];
		//lauch kernel : launch threads to update frontier_len, visited and frontier in gpu local mem
		s= clock();
		kernel<<<h_frontier[0], N>>>(d_adj_mat,N,d_visited,d_frontier, d_new_frontier);
		
		k2<<<1,1>>>(N, d_visited,d_frontier, d_new_frontier);
		e=clock();
		t+=double(e-s);

		hipMemcpy((void*) h_frontier, (void*) d_frontier, sizeof(int)*1, hipMemcpyDeviceToHost);
	}
	end = clock();
	cout<<"queue through put: "<< cn<<endl;
	cout << "parallel BFS uses " << double(end - start) << " us in total"<< endl;
	cout << "kernel launching and computing uses " <<t<<" us"<<endl;
	cout << "mem copy uses " <<double(end - start) - t<<" us"<<endl;

	return 0;
}
